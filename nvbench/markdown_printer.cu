#include "hip/hip_runtime.h"
/*
 *  Copyright 2021 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/markdown_printer.cuh>

#include <nvbench/benchmark_base.cuh>
#include <nvbench/device_manager.cuh>
#include <nvbench/state.cuh>
#include <nvbench/summary.cuh>

#include <nvbench/internal/markdown_table.cuh>

#include <fmt/color.h>
#include <fmt/format.h>

#include <functional>
#include <numeric>
#include <ostream>
#include <string>
#include <type_traits>
#include <vector>

namespace nvbench
{

void markdown_printer::do_print_device_info()
{
  fmt::memory_buffer buffer;
  fmt::format_to(std::back_inserter(buffer), "# Devices\n\n");

  const auto &device_mgr = nvbench::device_manager::get();
  const auto &devices = device_mgr.get_number_of_used_devices() > 0 ? device_mgr.get_used_devices()
                                                                    : device_mgr.get_devices();
  for (const auto &device : devices)
  {
    const auto [gmem_free, gmem_used] = device.get_global_memory_usage();

    fmt::format_to(std::back_inserter(buffer), "## [{}] `{}`\n", device.get_id(), device.get_name());
    fmt::format_to(std::back_inserter(buffer),
                   "* SM Version: {} (PTX Version: {})\n",
                   device.get_sm_version(),
                   device.get_ptx_version());
    fmt::format_to(std::back_inserter(buffer), "* Number of SMs: {}\n", device.get_number_of_sms());
    fmt::format_to(std::back_inserter(buffer),
                   "* SM Default Clock Rate: {} MHz\n",
                   device.get_sm_default_clock_rate() / 1000 / 1000);
    fmt::format_to(std::back_inserter(buffer),
                   "* Global Memory: {} MiB Free / {} MiB Total\n",
                   gmem_free / 1024 / 1024,
                   gmem_used / 1024 / 1024);
    fmt::format_to(std::back_inserter(buffer),
                   "* Global Memory Bus Peak: {} GB/sec ({}-bit DDR @{}MHz)\n",
                   device.get_global_memory_bus_bandwidth() / 1000 / 1000 / 1000,
                   device.get_global_memory_bus_width(),
                   device.get_global_memory_bus_peak_clock_rate() / 1000 / 1000);
    fmt::format_to(std::back_inserter(buffer),
                   "* Max Shared Memory: {} KiB/SM, {} KiB/Block\n",
                   device.get_shared_memory_per_sm() / 1024,
                   device.get_shared_memory_per_block() / 1024);
    fmt::format_to(std::back_inserter(buffer), "* L2 Cache Size: {} KiB\n", device.get_l2_cache_size() / 1024);
    fmt::format_to(std::back_inserter(buffer), "* Maximum Active Blocks: {}/SM\n", device.get_max_blocks_per_sm());
    fmt::format_to(std::back_inserter(buffer),
                   "* Maximum Active Threads: {}/SM, {}/Block\n",
                   device.get_max_threads_per_sm(),
                   device.get_max_threads_per_block());
    fmt::format_to(std::back_inserter(buffer),
                   "* Available Registers: {}/SM, {}/Block\n",
                   device.get_registers_per_sm(),
                   device.get_registers_per_block());
    fmt::format_to(std::back_inserter(buffer), "* ECC Enabled: {}\n", device.get_ecc_state() ? "Yes" : "No");
    fmt::format_to(std::back_inserter(buffer), "\n");
  }
  m_ostream << fmt::to_string(buffer);
}

void markdown_printer::do_print_log_preamble() { m_ostream << "# Log\n\n```\n"; }

void markdown_printer::do_print_log_epilogue() { m_ostream << "```\n\n"; }

void markdown_printer::do_log(nvbench::log_level level, const std::string &msg)
{
  const fmt::text_style no_style;
  const auto bg_bold = bg(fmt::color::black) | fmt::emphasis::bold;

  const auto run_color  = bg_bold | fg(fmt::color::white);
  const auto pass_color = bg_bold | fg(fmt::color::dark_green);
  const auto fail_color = bg_bold | fg(fmt::color::red);
  const auto skip_color = bg_bold | fg(fmt::color::steel_blue);
  const auto warn_color = bg_bold | fg(fmt::rgb{160, 135, 0}); // yellow
  const auto info_color = bg_bold | fg(fmt::color::light_gray);

  std::string tag;
  switch (level)
  {
    case log_level::run:
      tag = fmt::format(m_color ? run_color : no_style, "{:<5}", "Run:");
      break;
    case log_level::pass:
      tag = fmt::format(m_color ? pass_color : no_style, "{:<5}", "Pass:");
      break;
    case log_level::fail:
      tag = fmt::format(m_color ? fail_color : no_style, "{:<5}", "Fail:");
      break;
    case log_level::skip:
      tag = fmt::format(m_color ? skip_color : no_style, "{:<5}", "Skip:");
      break;
    case log_level::warn:
      tag = fmt::format(m_color ? warn_color : no_style, "{:<5}", "Warn:");
      break;
    case log_level::info:
      tag = fmt::format(m_color ? info_color : no_style, "{:<5}", "Info:");
      break;
  }

  // Flush each time; this is the only user-visible indication that a benchmark
  // is running.
  m_ostream << tag << " " << msg << std::endl;
}

void markdown_printer::do_log_run_state(const nvbench::state &exec_state)
{
  if (m_total_state_count == 0)
  { // No progress info
    this->log(nvbench::log_level::run, exec_state.get_short_description(m_color));
  }
  else
  { // Add progress
    this->log(nvbench::log_level::run,
              fmt::format("[{}/{}] {}",
                          m_completed_state_count + 1,
                          m_total_state_count,
                          exec_state.get_short_description(m_color)));
  }
}

void markdown_printer::do_print_benchmark_list(const printer_base::benchmark_vector &benches)
{
  if (benches.empty())
  {
    return;
  }

  fmt::memory_buffer buffer;
  fmt::format_to(std::back_inserter(buffer), "# Benchmarks\n\n");
  std::size_t benchmark_id{0};
  for (const auto &bench_ptr : benches)
  {
    const auto &axes              = bench_ptr->get_axes().get_axes();
    const std::size_t num_configs = bench_ptr->get_config_count();

    fmt::format_to(std::back_inserter(buffer),
                   "## [{}] `{}` ({} configurations)\n\n",
                   benchmark_id++,
                   bench_ptr->get_name(),
                   num_configs);

    fmt::format_to(std::back_inserter(buffer), "### Axes\n\n");
    for (const auto &axis_ptr : axes)
    {
      std::string flags_str(axis_ptr->get_flags_as_string());
      if (!flags_str.empty())
      {
        flags_str = fmt::format(" [{}]", flags_str);
      }
      fmt::format_to(std::back_inserter(buffer),
                     "* `{}` : {}{}\n",
                     axis_ptr->get_name(),
                     axis_ptr->get_type_as_string(),
                     flags_str);

      const std::size_t num_vals = axis_ptr->get_size();
      for (std::size_t i = 0; i < num_vals; ++i)
      {
        std::string desc = axis_ptr->get_description(i);
        if (!desc.empty())
        {
          desc = fmt::format(" ({})", desc);
        }
        fmt::format_to(std::back_inserter(buffer), "  * `{}`{}\n", axis_ptr->get_input_string(i), desc);
      } // end foreach value
    }   // end foreach axis
    fmt::format_to(std::back_inserter(buffer), "\n");
  } // end foreach bench

  m_ostream << fmt::to_string(buffer);
}

void markdown_printer::do_print_benchmark_results(const printer_base::benchmark_vector &benches)
{
  auto format_visitor = [](const auto &v) {
    using T = std::decay_t<decltype(v)>;
    if constexpr (std::is_same_v<T, nvbench::float64_t>)
    {
      return fmt::format("{:.5g}", v);
    }
    else if constexpr (std::is_same_v<T, std::string>)
    {
      return v;
    }

    // warning C4702: unreachable code
    // This is a future-proofing fallback that's currently unused.
    NVBENCH_MSVC_PUSH_DISABLE_WARNING(4702)
    return fmt::format("{}", v);
  };
  NVBENCH_MSVC_POP_WARNING()

  // Start printing benchmarks
  fmt::memory_buffer buffer;
  fmt::format_to(std::back_inserter(buffer), "# Benchmark Results\n");

  for (const auto &bench_ptr : benches)
  {
    const auto &bench   = *bench_ptr;
    const auto &devices = bench.get_devices();
    const auto &axes    = bench.get_axes();

    fmt::format_to(std::back_inserter(buffer), "\n## {}\n", bench.get_name());

    // Do a single pass when no devices are specified. This happens for
    // benchmarks with `cpu` exec_tags.
    const std::size_t num_device_passes = devices.empty() ? 1 : devices.size();
    for (std::size_t device_pass = 0; device_pass < num_device_passes; ++device_pass)
    {
      std::optional<nvbench::device_info> device = devices.empty()
                                                     ? std::nullopt
                                                     : std::make_optional(devices[device_pass]);

      if (device)
      {
        fmt::format_to(std::back_inserter(buffer), "\n### [{}] {}\n\n", device->get_id(), device->get_name());
      }

      std::size_t row = 0;
      nvbench::internal::markdown_table table{m_color};

      for (const auto &cur_state : bench.get_states())
      {
        if (cur_state.is_skipped())
        {
          continue;
        }

        if (cur_state.get_device() == device)
        {
          const auto &axis_values = cur_state.get_axis_values();
          for (const auto &name : axis_values.get_names())
          {
            // Handle power-of-two int64 axes differently:
            if (axis_values.get_type(name) == named_values::type::int64 &&
                axes.get_int64_axis(name).is_power_of_two())
            {
              const nvbench::int64_t value    = axis_values.get_int64(name);
              const nvbench::int64_t exponent = int64_axis::compute_log2(value);
              table.add_cell(row, name, name, fmt::format("2^{} = {}", exponent, value));
            }
            else
            {
              std::string value = std::visit(format_visitor, axis_values.get_value(name));
              table.add_cell(row, name + "_axis", name, std::move(value));
            }
          }

          for (const auto &summ : cur_state.get_summaries())
          {
            if (summ.has_value("hide"))
            {
              continue;
            }
            const std::string &tag    = summ.get_tag();
            const std::string &header = summ.has_value("name") ? summ.get_string("name") : tag;

            std::string hint = summ.has_value("hint") ? summ.get_string("hint") : std::string{};
            if (hint == "duration")
            {
              table.add_cell(row, tag, header, this->do_format_duration(summ));
            }
            else if (hint == "item_rate")
            {
              table.add_cell(row, tag, header, this->do_format_item_rate(summ));
            }
            else if (hint == "bytes")
            {
              table.add_cell(row, tag, header, this->do_format_bytes(summ));
            }
            else if (hint == "byte_rate")
            {
              table.add_cell(row, tag, header, this->do_format_byte_rate(summ));
            }
            else if (hint == "sample_size")
            {
              table.add_cell(row, tag, header, this->do_format_sample_size(summ));
            }
            else if (hint == "percentage")
            {
              table.add_cell(row, tag, header, this->do_format_percentage(summ));
            }
            else
            {
              table.add_cell(row, tag, header, this->do_format_default(summ));
            }
          }
          row++;
        }
      }

      auto table_str = table.to_string();
      fmt::format_to(std::back_inserter(buffer),
                     "{}",
                     table_str.empty() ? "No data -- check log.\n" : std::move(table_str));
    } // end foreach device_pass
  }

  m_ostream << fmt::to_string(buffer);
}

std::string markdown_printer::do_format_default(const summary &data)
{
  auto format_visitor = [](const auto &v) {
    using T = std::decay_t<decltype(v)>;
    if constexpr (std::is_same_v<T, nvbench::float64_t>)
    {
      return fmt::format("{:.5g}", v);
    }
    else if constexpr (std::is_same_v<T, std::string>)
    {
      return v;
    }

    // warning C4702: unreachable code
    // This is a future-proofing fallback that's currently unused.
    NVBENCH_MSVC_PUSH_DISABLE_WARNING(4702)
    return fmt::format("{}", v);
  };
  NVBENCH_MSVC_POP_WARNING()

  return std::visit(format_visitor, data.get_value("value"));
}

std::string markdown_printer::do_format_duration(const summary &data)
{
  const auto seconds = data.get_float64("value");
  if (seconds >= 1.) // 1+ sec
  {
    return fmt::format("{:0.3f} s", seconds);
  }
  else if (seconds >= 1e-3) // 1+ ms.
  {
    return fmt::format("{:0.3f} ms", seconds * 1e3);
  }
  else if (seconds >= 1e-6) // 1+ us.
  {
    return fmt::format("{:0.3f} us", seconds * 1e6);
  }
  else
  {
    return fmt::format("{:0.3f} ns", seconds * 1e9);
  }
}

std::string markdown_printer::do_format_item_rate(const summary &data)
{
  const auto items_per_second = data.get_float64("value");
  if (items_per_second >= 1e15)
  {
    return fmt::format("{:0.3f}P", items_per_second * 1e-15);
  }
  else if (items_per_second >= 1e12)
  {
    return fmt::format("{:0.3f}T", items_per_second * 1e-12);
  }
  else if (items_per_second >= 1e9)
  {
    return fmt::format("{:0.3f}G", items_per_second * 1e-9);
  }
  else if (items_per_second >= 1e6)
  {
    return fmt::format("{:0.3f}M", items_per_second * 1e-6);
  }
  else if (items_per_second >= 1e3)
  {
    return fmt::format("{:0.3f}K", items_per_second * 1e-3);
  }
  else
  {
    return fmt::format("{:0.3f}", items_per_second);
  }
}

std::string markdown_printer::do_format_bytes(const summary &data)
{
  const auto bytes = static_cast<nvbench::float64_t>(data.get_int64("value"));
  if (bytes >= 1024. * 1024. * 1024.) // 1 GiB
  {
    return fmt::format("{:0.3f} GiB", bytes / (1024. * 1024. * 1024.));
  }
  else if (bytes >= 1024. * 1024.) // 1 MiB
  {
    return fmt::format("{:0.3f} MiB", bytes / (1024. * 1024.));
  }
  else if (bytes >= 1024) // 1 KiB.
  {
    return fmt::format("{:0.3f} KiB", bytes / 1024.);
  }
  else
  {
    return fmt::format("{:0.3f} B", static_cast<nvbench::float64_t>(bytes));
  }
}

std::string markdown_printer::do_format_byte_rate(const summary &data)
{
  const auto bytes_per_second = data.get_float64("value");
  if (bytes_per_second >= 1e15)
  {
    return fmt::format("{:0.3f} PB/s", bytes_per_second * 1e-15);
  }
  else if (bytes_per_second >= 1e12)
  {
    return fmt::format("{:0.3f} TB/s", bytes_per_second * 1e-12);
  }
  else if (bytes_per_second >= 1e9)
  {
    return fmt::format("{:0.3f} GB/s", bytes_per_second * 1e-9);
  }
  else if (bytes_per_second >= 1e6)
  {
    return fmt::format("{:0.3f} MB/s", bytes_per_second * 1e-6);
  }
  else if (bytes_per_second >= 1e3)
  {
    return fmt::format("{:0.3f} KB/s", bytes_per_second * 1e-3);
  }
  else
  {
    return fmt::format("{:0.3f} B/s", bytes_per_second);
  }
}

std::string markdown_printer::do_format_sample_size(const summary &data)
{
  const auto count = data.get_int64("value");
  return fmt::format("{}x", count);
}

std::string markdown_printer::do_format_percentage(const summary &data)
{
  const auto percentage = data.get_float64("value");
  return fmt::format("{:.2f}%", percentage * 100.);
}

} // namespace nvbench
