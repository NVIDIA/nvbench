/*
 *  Copyright 2024 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/nvbench.cuh>

#include <algorithm>
#include <cstdlib>
#include <cstdio>

/******************************************************************************
 * Test having global state that is initialized and finalized via RAII.
 *****************************************************************************/

struct raii
{
  const char m_ref_data[6];
  char *m_data;
  bool m_cuda;

  const char *m_outer_data;
  bool m_outer_cuda;

  explicit raii(bool cuda, char *outer_data = nullptr, bool outer_cuda = false)
      : m_ref_data{'a', 'b', 'c', '1', '2', '3'}
      , m_data(nullptr)
      , m_cuda(cuda)
      , m_outer_data(outer_data)
      , m_outer_cuda(outer_cuda)
  {
    if (m_cuda)
    {
      printf("(%p) RAII test: allocating device memory\n", this);
      NVBENCH_CUDA_CALL(hipMalloc(&m_data, 6));
      NVBENCH_CUDA_CALL(hipMemcpy(m_data, m_ref_data, 6, hipMemcpyHostToDevice));
    }
    else
    {
      printf("(%p) RAII test: allocating host memory\n", this);
      m_data = new char[6];
      std::copy(m_ref_data, m_ref_data + 6, m_data);
    }
  }

  ~raii()
  {
    this->verify();
    if (m_cuda)
    {
      printf("(%p) RAII test: invalidating device memory\n", this);
      NVBENCH_CUDA_CALL(hipMemset(m_data, 0, 6));
      printf("(%p) RAII test: freeing device memory\n", this);
      NVBENCH_CUDA_CALL(hipFree(m_data));
    }
    else
    {
      printf("(%p) RAII test: invalidating host memory\n", this);
      std::fill(m_data, m_data + 6, '\0');
      printf("(%p) RAII test: freeing host memory\n", this);
      delete[] m_data;
    }
  }

  void verify() noexcept
  {
    printf("(%p) RAII test: verifying instance state\n", this);
    this->verify(m_cuda, m_data);
    if (m_outer_data)
    {
      printf("(%p) RAII test: verifying outer state\n", this);
      this->verify(m_outer_cuda, m_outer_data);
    }
  }

  void verify(bool cuda, const char *data) noexcept
  {
    if (cuda)
    {
      char test_data[6];
      NVBENCH_CUDA_CALL(hipMemcpy(test_data, data, 6, hipMemcpyDeviceToHost));
      if (strncmp(test_data, m_ref_data, 6) != 0)
      {
        printf("(%p) RAII test failed: device data mismatch\n", this);
        std::exit(1);
      }
    }
    else
    {
      if (strncmp(data, m_ref_data, 6) != 0)
      {
        printf("(%p) RAII test failed: host data mismatch\n", this);
        std::exit(1);
      }
    }
  }
};

// These will be destroyed in the opposite order in which they are created:

#undef NVBENCH_MAIN_INITIALIZE_CUSTOM_PRE
#define NVBENCH_MAIN_INITIALIZE_CUSTOM_PRE(argc, argv) raii raii_outer(false);

#undef NVBENCH_MAIN_INITIALIZE_CUSTOM_POST
#define NVBENCH_MAIN_INITIALIZE_CUSTOM_POST(argc, argv)                                            \
  [[maybe_unused]] raii raii_inner(true, raii_outer.m_data, raii_outer.m_cuda);

NVBENCH_MAIN
