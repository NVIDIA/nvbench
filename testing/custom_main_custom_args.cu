#include "hip/hip_runtime.h"
/*
 *  Copyright 2024 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/nvbench.cuh>
#include "nvbench/cuda_call.cuh"

/******************************************************************************
 * Install custom parser.
 * sSee <nvbench/main.cuh> for more details.
 ******************************************************************************/

//
// Step 1: Define a custom argument handler that accepts a vector of strings.
//          - This handler should modify the vector in place to remove any custom
//            arguments it handles. NVbench will then parse the remaining arguments.
//          - The handler should also update any application state needed to handle
//            the custom arguments.
//

// User code to handle a specific argument:
void handle_my_custom_arg();

// NVBench hook for modiifying the command line arguments before parsing:
void custom_arg_handler(std::vector<std::string> &args)
{
  // Handle and remove "--my-custom-arg"
  if (auto it = std::find(args.begin(), args.end(), "--my-custom-arg"); it != args.end())
  {
    handle_my_custom_arg();
    args.erase(it);
  }
}

//
// Step 2: Install the custom argument handler.
//         - This is done by defining a macro that invokes the custom argument handler.
//

// Install the custom argument handler:
// Either define this before any NVBench headers are included, or undefine and redefine:
#undef NVBENCH_MAIN_CUSTOM_ARGS_HANDLER
#define NVBENCH_MAIN_CUSTOM_ARGS_HANDLER(args) custom_arg_handler(args)

// Step 3: Define `main`
//
// After installing the custom argument handler, define the main function using:
//
// ```
// NVBENCH_MAIN
// ```
//
// Here, this is done at the end of this file.

/******************************************************************************
 * Unit test verification:
 ******************************************************************************/

// Track whether the args are found / handled.
bool h_custom_arg_found             = false;
bool h_handled_on_device            = false;
__device__ bool d_custom_arg_found  = false;
__device__ bool d_handled_on_device = false;

// Copy host values to device:
void copy_host_state_to_device()
{
  NVBENCH_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_custom_arg_found), &h_custom_arg_found, sizeof(bool)));
  NVBENCH_CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_handled_on_device), &h_handled_on_device, sizeof(bool)));
}

// Copy device values to host:
void copy_device_state_to_host()
{
  NVBENCH_CUDA_CALL(hipMemcpyFromSymbol(&h_custom_arg_found, HIP_SYMBOL(d_custom_arg_found), sizeof(bool)));
  NVBENCH_CUDA_CALL(hipMemcpyFromSymbol(&h_handled_on_device, HIP_SYMBOL(d_handled_on_device), sizeof(bool)));
}

void handle_my_custom_arg()
{
  h_custom_arg_found = true;
  copy_host_state_to_device();
}

void verify()
{
  copy_device_state_to_host();
  if (!h_custom_arg_found)
  {
    throw std::runtime_error("Custom argument not detected.");
  }
  if (!h_handled_on_device)
  {
    throw std::runtime_error("Custom argument not handled on device.");
  }
}

// Install a verification check to ensure the custom argument was handled.
// Use the `PRE` finalize hook to ensure we check device state before resetting the context.
#undef NVBENCH_MAIN_FINALIZE_CUSTOM_PRE
#define NVBENCH_MAIN_FINALIZE_CUSTOM_PRE() verify()

// Simple kernel/benchmark to make sure that the handler can successfully modify CUDA state:
__global__ void kernel()
{
  if (d_custom_arg_found)
  {
    d_handled_on_device = true;
  }
}
void bench(nvbench::state &state)
{
  state.exec([](nvbench::launch &) { kernel<<<1, 1>>>(); });
}
NVBENCH_BENCH(bench);

// Define the customized main function:
NVBENCH_MAIN
