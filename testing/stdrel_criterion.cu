#include "hip/hip_runtime.h"
/*
 *  Copyright 2023 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/detail/stdrel_criterion.cuh>
#include <nvbench/stopping_criterion.cuh>
#include <nvbench/types.cuh>

#include "test_asserts.cuh"

#include <vector>
#include <random>
#include <numeric>

void test_const()
{
  nvbench::criterion_params params;
  nvbench::detail::stdrel_criterion criterion;

  criterion.initialize(params);
  for (int i = 0; i < 5; i++)
  { // nvbench wants at least 5 to compute the standard deviation
    criterion.add_measurement(42.0);
  }
  ASSERT(criterion.is_finished());
}

std::vector<double> generate(double mean, double rel_std_dev, int size)
{
  static std::mt19937::result_type seed = 0;
  std::mt19937 gen(seed++);
  std::vector<nvbench::float64_t> v(static_cast<std::size_t>(size));
  std::normal_distribution<nvbench::float64_t> dist(mean, mean * rel_std_dev);
  std::generate(v.begin(), v.end(), [&]{ return dist(gen); });
  return v;
}

void test_stdrel()
{
  const nvbench::int64_t size = 10;
  const nvbench::float64_t mean = 42.0;
  const nvbench::float64_t max_noise = 0.1;

  nvbench::criterion_params params;
  params.set_float64("max-noise", max_noise);

  nvbench::detail::stdrel_criterion criterion;
  criterion.initialize(params);

  for (nvbench::float64_t measurement: generate(mean, max_noise / 2, size))
  {
    criterion.add_measurement(measurement);
  }
  ASSERT(criterion.is_finished());

  params.set_float64("max-noise", max_noise);
  criterion.initialize(params);

  for (nvbench::float64_t measurement: generate(mean, max_noise * 2, size))
  {
    criterion.add_measurement(measurement);
  }
  ASSERT(!criterion.is_finished());
}

int main()
{
  test_const();
  test_stdrel();
}
