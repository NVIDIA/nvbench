/*
 *  Copyright 2021 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/type_axis.cuh>
#include <nvbench/types.cuh>

#include <fmt/format.h>

#include "test_asserts.cuh"

void test_empty()
{
  nvbench::type_axis axis("Basic", 0);

  ASSERT(axis.get_name() == "Basic");
  ASSERT(axis.get_axis_index() == 0);
  ASSERT(axis.get_type() == nvbench::axis_type::type);
  ASSERT(axis.get_size() == 0);

  axis.set_inputs<nvbench::type_list<>>();

  ASSERT(axis.get_size() == 0);

  const auto clone_base = axis.clone();
  ASSERT(clone_base.get() != nullptr);
  const auto *clone = dynamic_cast<const nvbench::type_axis *>(clone_base.get());
  ASSERT(clone != nullptr);

  ASSERT(clone->get_name() == "Basic");
  ASSERT(clone->get_axis_index() == 0);
  ASSERT(clone->get_type() == nvbench::axis_type::type);
  ASSERT(clone->get_size() == 0);
}

void test_single()
{
  nvbench::type_axis axis("Single", 0);
  axis.set_inputs<nvbench::type_list<nvbench::int32_t>>();

  ASSERT(axis.get_name() == "Single");
  ASSERT(axis.get_size() == 1);
  ASSERT(axis.get_input_string(0) == "I32");
  ASSERT(axis.get_description(0) == "int32_t");
  ASSERT(axis.get_is_active("I32") == true);
  ASSERT(axis.get_is_active(0) == true);

  auto clone_base = axis.clone();
  ASSERT(clone_base.get() != nullptr);
  auto *clone = dynamic_cast<nvbench::type_axis *>(clone_base.get());
  ASSERT(clone != nullptr);

  ASSERT(clone->get_name() == "Single");
  ASSERT(clone->get_size() == 1);
  ASSERT(clone->get_input_string(0) == "I32");
  ASSERT(clone->get_description(0) == "int32_t");
  ASSERT(clone->get_is_active("I32") == true);
  ASSERT(clone->get_is_active(0) == true);

  clone->set_active_inputs({});
  ASSERT(clone->get_name() == "Single");
  ASSERT(clone->get_size() == 1);
  ASSERT(clone->get_input_string(0) == "I32");
  ASSERT(clone->get_description(0) == "int32_t");
  ASSERT(clone->get_is_active("I32") == false);
  ASSERT(clone->get_is_active(0) == false);
  // The original property should not be modified:
  ASSERT(axis.get_is_active("I32") == true);
  ASSERT(axis.get_is_active(0) == true);

  clone->set_active_inputs({"I32"});
  ASSERT(clone->get_name() == "Single");
  ASSERT(clone->get_size() == 1);
  ASSERT(clone->get_input_string(0) == "I32");
  ASSERT(clone->get_description(0) == "int32_t");
  ASSERT(clone->get_is_active("I32") == true);
  ASSERT(clone->get_is_active(0) == true);
  // The original property should not be modified:
  ASSERT(axis.get_is_active("I32") == true);
  ASSERT(axis.get_is_active(0) == true);

  ASSERT_THROWS_ANY(clone->set_active_inputs({"NotAValidEntry"}));
}

void test_several()
{
  nvbench::type_axis axis("Several", 0);
  axis.set_inputs<nvbench::type_list<nvbench::int32_t, nvbench::float64_t, bool>>();

  ASSERT(axis.get_name() == "Several");
  ASSERT(axis.get_size() == 3);
  ASSERT(axis.get_input_string(0) == "I32");
  ASSERT(axis.get_description(0) == "int32_t");
  ASSERT(axis.get_is_active(0) == true);
  ASSERT(axis.get_is_active("I32") == true);
  ASSERT(axis.get_input_string(1) == "F64");
  ASSERT(axis.get_description(1) == "double");
  ASSERT(axis.get_is_active(1) == true);
  ASSERT(axis.get_is_active("F64") == true);
  ASSERT(axis.get_input_string(2) == "bool");
  ASSERT(axis.get_description(2) == "");
  ASSERT(axis.get_is_active(2) == true);
  ASSERT(axis.get_is_active("bool") == true);

  auto clone_base = axis.clone();
  ASSERT(clone_base.get() != nullptr);
  auto *clone = dynamic_cast<nvbench::type_axis *>(clone_base.get());
  ASSERT(clone != nullptr);

  ASSERT(clone->get_name() == "Several");
  ASSERT(clone->get_size() == 3);
  ASSERT(clone->get_input_string(0) == "I32");
  ASSERT(clone->get_description(0) == "int32_t");
  ASSERT(clone->get_is_active(0) == true);
  ASSERT(clone->get_is_active("I32") == true);
  ASSERT(clone->get_input_string(1) == "F64");
  ASSERT(clone->get_description(1) == "double");
  ASSERT(clone->get_is_active(1) == true);
  ASSERT(clone->get_is_active("F64") == true);
  ASSERT(clone->get_input_string(2) == "bool");
  ASSERT(clone->get_description(2) == "");
  ASSERT(clone->get_is_active(2) == true);
  ASSERT(clone->get_is_active("bool") == true);

  clone->set_active_inputs({"I32", "bool"});
  ASSERT(clone->get_name() == "Several");
  ASSERT(clone->get_size() == 3);
  ASSERT(clone->get_input_string(0) == "I32");
  ASSERT(clone->get_description(0) == "int32_t");
  ASSERT(clone->get_is_active(0) == true);
  ASSERT(clone->get_is_active("I32") == true);
  ASSERT(clone->get_input_string(1) == "F64");
  ASSERT(clone->get_description(1) == "double");
  ASSERT(clone->get_is_active(1) == false);
  ASSERT(clone->get_is_active("F64") == false);
  ASSERT(clone->get_input_string(2) == "bool");
  ASSERT(clone->get_description(2) == "");
  ASSERT(clone->get_is_active(2) == true);
  ASSERT(clone->get_is_active("bool") == true);

  // The cloned axis should not change:
  ASSERT(axis.get_name() == "Several");
  ASSERT(axis.get_size() == 3);
  ASSERT(axis.get_input_string(0) == "I32");
  ASSERT(axis.get_description(0) == "int32_t");
  ASSERT(axis.get_is_active(0) == true);
  ASSERT(axis.get_is_active("I32") == true);
  ASSERT(axis.get_input_string(1) == "F64");
  ASSERT(axis.get_description(1) == "double");
  ASSERT(axis.get_is_active(1) == true);
  ASSERT(axis.get_is_active("F64") == true);
  ASSERT(axis.get_input_string(2) == "bool");
  ASSERT(axis.get_description(2) == "");
  ASSERT(axis.get_is_active(2) == true);
  ASSERT(axis.get_is_active("bool") == true);
}

void test_get_type_index()
{
  nvbench::type_axis axis("GetIndexTest", 0);
  axis
    .set_inputs<nvbench::type_list<nvbench::int8_t, nvbench::uint16_t, nvbench::float32_t, bool>>();

  ASSERT(axis.get_type_index("I8") == 0);
  ASSERT(axis.get_type_index("U16") == 1);
  ASSERT(axis.get_type_index("F32") == 2);
  ASSERT(axis.get_type_index("bool") == 3);

  const auto clone_base = axis.clone();
  ASSERT(clone_base.get() != nullptr);
  const auto *clone = dynamic_cast<const nvbench::type_axis *>(clone_base.get());
  ASSERT(clone != nullptr);

  ASSERT(clone->get_type_index("I8") == 0);
  ASSERT(clone->get_type_index("U16") == 1);
  ASSERT(clone->get_type_index("F32") == 2);
  ASSERT(clone->get_type_index("bool") == 3);
}

int main()
{
  test_empty();
  test_single();
  test_several();
  test_get_type_index();
}
