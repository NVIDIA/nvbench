#include "hip/hip_runtime.h"
/*
 *  Copyright 2021 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/nvbench.cuh>

// Grab some testing kernels from NVBench:
#include <nvbench/test_kernels.cuh>

// Thrust vectors simplify memory management:
#include <thrust/device_vector.h>

// std::enable_if_t
#include <type_traits>

//==============================================================================
// `runtime_skip` demonstrates how to skip benchmarks at runtime.
//
// Two parameter axes are swept (see axes.cu), but some configurations are
// skipped by calling `state.skip` with a skip reason string. This reason
// is printed to the log and captured in JSON output.
void runtime_skip(nvbench::state &state)
{
  const auto duration = state.get_float64("Duration");
  const auto kramble  = state.get_string("Kramble");

  // Skip Baz benchmarks with < 0.8 ms duration.
  if (kramble == "Baz" && duration < 0.8e-3)
  {
    state.skip("Short 'Baz' benchmarks are skipped.");
    return;
  }

  // Skip Foo benchmarks with > 0.3 ms duration.
  if (kramble == "Foo" && duration > 0.3e-3)
  {
    state.skip("Long 'Foo' benchmarks are skipped.");
    return;
  }

  // Run all others:
  state.exec([duration](nvbench::launch &launch) {
    nvbench::sleep_kernel<<<1, 1, 0, launch.get_stream()>>>(duration);
  });
}
NVBENCH_BENCH(runtime_skip)
  // 0, 0.25, 0.5, 0.75, and 1.0 milliseconds
  .add_float64_axis("Duration",
                    nvbench::range(0.,
                                   1.1e-3, // .1e-3 slop for fp precision
                                   0.25e-3))
  .add_string_axis("Kramble", {"Foo", "Bar", "Baz"});

//==============================================================================
// `skip_overload` demonstrates how to skip benchmarks at compile-time via
// overload resolution.
//
// Two type axes are swept, but configurations where InputType == OutputType are
// skipped.
template <typename InputType, typename OutputType>
void skip_overload(nvbench::state &state, nvbench::type_list<InputType, OutputType>)
{
  // This is a contrived example that focuses on the skip overloads, so this is
  // just a sleep kernel:
  state.exec(
    [](nvbench::launch &launch) { nvbench::sleep_kernel<<<1, 1, 0, launch.get_stream()>>>(1e-3); });
}
// Overload of skip_overload that is called when InputType == OutputType.
template <typename T>
void skip_overload(nvbench::state &state, nvbench::type_list<T, T>)
{
  state.skip("InputType == OutputType.");
}
// The same type_list is used for both inputs/outputs.
using sst_types = nvbench::type_list<nvbench::int32_t, nvbench::int64_t>;
// Setup benchmark:
NVBENCH_BENCH_TYPES(skip_overload, NVBENCH_TYPE_AXES(sst_types, sst_types))
  .set_type_axes_names({"In", "Out"});

//==============================================================================
// `skip_sfinae` demonstrates how to skip benchmarks at compile-time using
// SFINAE to handle more complex skip conditions.
//
// Two type axes are swept, but configurations where sizeof(InputType) >
// sizeof(OutputType) are skipped.

// Enable this overload if InputType is not larger than OutputType
template <typename InputType, typename OutputType>
std::enable_if_t<(sizeof(InputType) <= sizeof(OutputType)), void>
skip_sfinae(nvbench::state &state, nvbench::type_list<InputType, OutputType>)
{
  // This is a contrived example that focuses on the skip overloads, so this is
  // just a sleep kernel:
  state.exec(
    [](nvbench::launch &launch) { nvbench::sleep_kernel<<<1, 1, 0, launch.get_stream()>>>(1e-3); });
}
// Enable this overload if InputType is larger than OutputType
template <typename InputType, typename OutputType>
std::enable_if_t<(sizeof(InputType) > sizeof(OutputType)), void>
skip_sfinae(nvbench::state &state, nvbench::type_list<InputType, OutputType>)
{
  state.skip("sizeof(InputType) > sizeof(OutputType).");
}
// The same type_list is used for both inputs/outputs.
using sn_types =
  nvbench::type_list<nvbench::int8_t, nvbench::int16_t, nvbench::int32_t, nvbench::int64_t>;
// Setup benchmark:
NVBENCH_BENCH_TYPES(skip_sfinae, NVBENCH_TYPE_AXES(sn_types, sn_types))
  .set_type_axes_names({"In", "Out"});
