/*
 *  Copyright 2022 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/nvbench.cuh>

// Grab some testing kernels from NVBench:
#include <nvbench/test_kernels.cuh>

// Thrust vectors simplify memory management:
#include <thrust/device_vector.h>

// A function to benchmark but does not expose an explicit stream argument.
void copy(int32_t *input, int32_t *output, std::size_t const num_values)
{
  nvbench::copy_kernel<<<256, 256>>>(input, output, num_values);
}

// `stream_bench` copies a 64 MiB buffer of int32_t on a CUDA stream specified
// by the user.
//
// By default, NVBench creates and provides an explicit stream via
// `launch::get_stream()` to pass to every stream-ordered operation. Sometimes
// it is inconvenient or impossible to specify an explicit CUDA stream to every
// stream-ordered operation. In this case, users may specify a target stream via
// `state::set_cuda_stream`. It is assumed that all work of interest executes on
// or synchronizes with this stream.
void stream_bench(nvbench::state &state)
{
  // Allocate input data:
  const std::size_t num_values = 64 * 1024 * 1024 / sizeof(nvbench::int32_t);
  thrust::device_vector<nvbench::int32_t> input(num_values);
  thrust::device_vector<nvbench::int32_t> output(num_values);

  // Set CUDA default stream as the target stream. Note the default stream
  // is non-owning.
  hipStream_t default_stream = 0;
  state.set_cuda_stream(nvbench::make_cuda_stream_view(default_stream));

  state.exec([&input, &output, num_values](nvbench::launch &) {
    (void) num_values; // clang thinks this is unused...
    copy(thrust::raw_pointer_cast(input.data()),
         thrust::raw_pointer_cast(output.data()),
         num_values);
  });
}
NVBENCH_BENCH(stream_bench);
