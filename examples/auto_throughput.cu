#include "hip/hip_runtime.h"
/*
 *  Copyright 2021 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 with the LLVM exception
 *  (the "License"); you may not use this file except in compliance with
 *  the License.
 *
 *  You may obtain a copy of the License at
 *
 *      http://llvm.org/foundation/relicensing/LICENSE.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <nvbench/nvbench.cuh>

// Thrust vectors simplify memory management:
#include <thrust/device_vector.h>

template <int ItemsPerThread>
__global__ void kernel(std::size_t stride,
                       std::size_t elements,
                       const nvbench::int32_t *__restrict__ in,
                       nvbench::int32_t *__restrict__ out)
{
  const std::size_t tid  = threadIdx.x + blockIdx.x * blockDim.x;
  const std::size_t step = gridDim.x * blockDim.x;

  for (std::size_t i = stride * tid; i < stride * elements; i += stride * step)
  {
    for (int j = 0; j < ItemsPerThread; j++)
    {
      const auto read_id  = (ItemsPerThread * i + j) % elements;
      const auto write_id = tid + j * elements;
      out[write_id]       = in[read_id];
    }
  }
}

// `throughput_bench` copies a 128 MiB buffer of int32_t, and reports throughput
// and cache hit rates.
//
// Calling state.collect_*() enables particular metric collection if nvbench
// was build with CUPTI support (CMake option: -DNVBench_ENABLE_CUPTI=ON).
template <int ItemsPerThread>
void throughput_bench(nvbench::state &state, nvbench::type_list<nvbench::enum_type<ItemsPerThread>>)
{
  // Allocate input data:
  const std::size_t stride   = static_cast<std::size_t>(state.get_int64("Stride"));
  const std::size_t elements = 128 * 1024 * 1024 / sizeof(nvbench::int32_t);
  thrust::device_vector<nvbench::int32_t> input(elements);
  thrust::device_vector<nvbench::int32_t> output(elements * ItemsPerThread);

  // Provide throughput information:
  state.add_element_count(elements, "Elements");
  state.collect_dram_throughput();
  state.collect_l1_hit_rates();
  state.collect_l2_hit_rates();
  state.collect_loads_efficiency();
  state.collect_stores_efficiency();

  const auto threads_in_block = 256;
  const auto blocks_in_grid =
    static_cast<int>((elements + threads_in_block - 1) / threads_in_block);

  state.exec([&](nvbench::launch &launch) {
    kernel<ItemsPerThread><<<blocks_in_grid, threads_in_block, 0, launch.get_stream()>>>(
      stride,
      elements,
      thrust::raw_pointer_cast(input.data()),
      thrust::raw_pointer_cast(output.data()));
  });
}

using items_per_thread = nvbench::enum_type_list<1, 2>;

NVBENCH_BENCH_TYPES(throughput_bench, NVBENCH_TYPE_AXES(items_per_thread))
  .add_int64_axis("Stride", nvbench::range(1, 4, 3));
